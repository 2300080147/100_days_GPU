#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    // Fill A and B with numbers
    for (int i = 0; i < N; ++i) {
        A[i] = i;
        B[i] = i * 2;
    }

    // Allocate memory on GPU
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy A and B to GPU
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;
    vectorAdd<<<gridsize, blocksize>>>(d_a, d_b, d_c, N);

    // Copy result back to CPU
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result of vector addition:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
